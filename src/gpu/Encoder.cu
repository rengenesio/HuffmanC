#include "hip/hip_runtime.h"
#include "huffman.h"


#define BLOCKS_PER_GRID		1
#define THREADS_PER_BLOCK	512
#define TOTAL_THREADS		(BLOCKS_PER_GRID * THREADS_PER_BLOCK)



void * threadTask(void *tid);
void initializeVars(void);
__global__ void frequencyErase (FREQUENCY *f);
__global__ void memoryToFrequency (SYMBOL *m, FREQUENCY *f, FILE_SIZE s);
__global__ void addFrequency (FREQUENCY *f, FREQUENCY *f_total);
void findEof (void);
void frequencyToNodeArray(void);
void treeToCode (void);
__global__ void fileCompressor(unsigned long long int *written_bits, BYTE *memory_out, CODIFICATION *codification, SYMBOL *cuda_memory_in, FILE_SIZE s, SIZE symbols);
void memoryToFile (void);


FILE *in = NULL;
FILE *out = NULL;
FILE *cod = NULL;

SYMBOL eof;
SYMBOL *memory_in, *cuda_memory_in;
FILE_SIZE size_in;
BYTE *memory_out, *cuda_memory_out;
SIZE symbols = 0;
SIZE total_codes = 0;
unsigned long long int *written_bits, *cuda_written_bits;
FREQUENCY *frequency, *frequency_total, *cuda_frequency, *cuda_frequency_total;
NODE_ARRAY *node_array;
CODIFICATION *codification;


unsigned char visited1 = 0;
unsigned char visited2 = 0;
unsigned char visited3 = 0;
unsigned char visited4 = 0;

unsigned char visited5 = 0;
unsigned char visited6 = 0;
unsigned char visited7 = 0;
unsigned char visited8 = 0;
unsigned char visited9 = 0;

pthread_mutex_t mutex;
pthread_mutex_t mutex2;
pthread_barrier_t barrier;


int main(int argc, char *argv[]) {
	unsigned int i;

	openFiles(&in, argv[1], "rb", &out, argv[2], "wb", &cod, argv[3], "wb");

//#ifdef DEBUG
	fprintf(stderr, "\nFILES:\n\nIn: %s\nOut: %s\nCB: %s\n", argv[1], argv[2], argv[3]);
//#endif
	memory_in = fileToMemory(in, &size_in);
//#ifdef DEBUG
	fprintf(stderr, "\n\nSIZE:\n%u bytes\n", size_in);
//#endif

	if(hipMalloc(&cuda_frequency, sizeof(FREQUENCY) * TOTAL_THREADS * POWER_BITS_CODIFICATION) != hipSuccess) { printf("Erro hipMalloc cuda_frequency\n"); return -1; }
	dim3 dim_grid(16, 16, 1);
	dim3 dim_block(32, 16);
	frequencyErase <<< dim_grid , dim_block  >>> (cuda_frequency);

//#ifdef DEBUG
	if((frequency = (FREQUENCY *) malloc(sizeof(FREQUENCY) * TOTAL_THREADS * POWER_BITS_CODIFICATION)) == NULL) { printf("Erro malloc frequency\n"); return -1; }
	if(hipMemcpy(frequency, cuda_frequency, sizeof(FREQUENCY) * TOTAL_THREADS * POWER_BITS_CODIFICATION, hipMemcpyDeviceToHost) != hipSuccess) { printf("Erro hipMemcpy cuda_frequency\n"); return -1; }

	fprintf(stderr, "\n\nFREQUENCY (IF NOT ERASED): symbol (frequency)\n\n");
	for(i = 0 ; i < TOTAL_THREADS * POWER_BITS_CODIFICATION ; i++)
		if(frequency[i])
			fprintf(stderr, "%u (%u)\n", i, frequency[i]);
//#endif

	if(hipMalloc(&cuda_memory_in, sizeof(SYMBOL) * size_in) != hipSuccess) { printf("Erro hipMalloc cuda_memory_in\n"); return -1; }
	if(hipMemcpy(cuda_memory_in, memory_in, sizeof(SYMBOL) * size_in, hipMemcpyHostToDevice) != hipSuccess) { printf("Erro hipMemcpy cuda_memory_in\n"); return -1; }
	memoryToFrequency <<< BLOCKS_PER_GRID , THREADS_PER_BLOCK >>> (cuda_memory_in, cuda_frequency, size_in);

	if(hipMalloc(&cuda_frequency_total, sizeof(FREQUENCY) * POWER_BITS_CODIFICATION) != hipSuccess) { printf("Erro hipMalloc cuda_frequency_total\n"); return -1; }
	addFrequency <<< 1 , POWER_BITS_CODIFICATION >>> (cuda_frequency, cuda_frequency_total);

//#ifdef DEBUG
	if((frequency_total = (FREQUENCY *) malloc(sizeof(FREQUENCY) * POWER_BITS_CODIFICATION)) == NULL) { printf("Erro malloc frequency_total\n"); return -1; }
	if(hipMemcpy(frequency_total, cuda_frequency_total, sizeof(FREQUENCY) * POWER_BITS_CODIFICATION, hipMemcpyDeviceToHost) != hipSuccess) { printf("Erro hipMemcpy cuda_frequency_total\n"); return -1; }
	fprintf(stderr, "\n\nFREQUENCY_TOTAL (IF NOT ERASED): symbol (frequency_total)\n\n");
			
	for(i = 0 ; i < POWER_BITS_CODIFICATION ; i++)
		if(frequency_total[i])
			fprintf(stderr, "%u (%u)\n", i, frequency_total[i]);
//#endif

	findEof();
	node_array = newNodeArray(symbols);
	codification = (CODIFICATION *) malloc(sizeof(CODIFICATION) * symbols);
	if(!codification) {
		fprintf(STDOUT, "Erro ao alocar variavel 'codification'\n");
		exit(-1);
	}

	frequencyToNodeArray();

//#ifdef DEBUG
	fprintf(stderr, "\n\nNODE ARRAY: symbol (frequency)\n\n");
	for(i = 0 ; i < node_array->size ; i++)
		fprintf(stderr, "%u (%u)\n", node_array->node[i].symbol, node_array->node[i].frequency);
//#endif

	huffmanEncode(node_array);
	treeToCode();

//#ifdef DEBUG
	fprintf(stderr, "\n\nCODIFICATION: symbol (size) code\n\n");
	for(unsigned int i = 0 ; i < symbols ; i++)
		fprintf(stderr, "i: %u - %x (%u) %s\n", i, codification[i].symbol, codification[i].size, codification[i].code);
//#endif

	codificationToFile(cod, codification, symbols, eof);

	if(hipMalloc(&cuda_written_bits, sizeof(unsigned long long int) * TOTAL_THREADS) != hipSuccess) { printf("Erro hipMalloc cuda_written_bits\n"); return -1; }
	if(hipMalloc(&cuda_memory_out, sizeof(BYTE) * TOTAL_THREADS * size_in) != hipSuccess) { printf("Erro hipMalloc cuda_memory_out\n"); return -1; }
	fileCompressor <<< BLOCKS_PER_GRID , THREADS_PER_BLOCK >>> (cuda_written_bits, cuda_memory_out, codification, cuda_memory_in, size_in, symbols);
	memoryToFile();

	return 0;
}

__global__ void frequencyErase (FREQUENCY *f) {
	f[(blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x) + (blockIdx.x * blockDim.x + threadIdx.x)] = 0;
}

__global__ void memoryToFrequency (SYMBOL *m, FREQUENCY *f, FILE_SIZE s) {
	unsigned int start, blocks;
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	blocks = s / TOTAL_THREADS;
	blocks = tid < (s % TOTAL_THREADS) ? blocks+1 : blocks;

	start = (s / TOTAL_THREADS) * tid;
	start = tid >= (s % TOTAL_THREADS) ? start + (s % TOTAL_THREADS) : start + tid;

	unsigned int i;
	for(i = 0 ; i < blocks ; i++)
		f[tid * POWER_BITS_CODIFICATION + m[start + i]]++;
}

__global__ void addFrequency (FREQUENCY *f, FREQUENCY *f_total) {
	unsigned int tid = threadIdx.x;
	f_total[tid] = 0;

	unsigned int i;
	for(i = 0 ; i < TOTAL_THREADS ; i++)
		f_total[tid] += f[(i * POWER_BITS_CODIFICATION) + tid];
}

void findEof (void) {
	symbols = 0;
	unsigned int eof_found = 0;

	unsigned int i;
	for(i = 0 ; i < POWER_BITS_CODIFICATION ; i++) {
		if(frequency_total[i] == 0) {
			if(!eof_found) {
				eof_found = 1;
				eof = i;
				symbols++;
				frequency_total[i]++;
				memset(memory_in + (size_in / SIZEOF_SYMBOL), eof, SIZEOF_SYMBOL);
				size_in += SIZEOF_SYMBOL;
			}
		}
		else
			symbols++;
	}
}

void frequencyToNodeArray (void) {
	unsigned int i;
	for(i = 0 ; i < POWER_BITS_CODIFICATION ; i++) {
		if(frequency_total[i]) {
			NODE *n = (NODE *) malloc(sizeof(NODE));
			newNode(n, i, frequency_total[i], NULL, NULL);
			insertSorted(node_array, n);
		}
	}
}

void treeToCode (void) {
	STACK *s = newStack(2 * POWER_BITS_CODIFICATION - 1);
	NODE *n = &node_array->node[0];
	SIZE codes = 0;

	unsigned char path[(int) POWER_BITS_CODIFICATION];
	memset(path, '\0', POWER_BITS_CODIFICATION);

	push(s, n);
	while(codes < symbols) {
		if(n->left) {
			if(!n->left->visited) {
				push(s, n);
				n->visited = 1;
				n = n->left;
				path[strlenU(path)] = '0';
			}
			else if(!n->right->visited) {
				push(s, n);
				n->visited = 1;
				n = n->right;
				path[strlenU(path)] = '1';
			}
			else {
				path[strlenU(path)-1] = '\0';
				n = pop(s);
			}
		}
		else {
			n->visited = 1;
			codification[codes].size = strlenU(path);
			codification[codes].symbol = n->symbol;
			codification[codes].code = (unsigned char *) malloc(sizeof(unsigned char) * (codification[codes].size + 1));
			if(!codification[codes].code) {
				fprintf(STDOUT, "Erro ao alocar variavel 'codification[%u].code'\n", codes);
				exit(-1);
			}
			memcpy(codification[codes].code, path, codification[codes].size + 1);
			path[strlenU(path)-1] = '\0';
			n = pop(s);
			codes++;
		}
	}
}

__global__ void fileCompressor(unsigned long long int *wb, BYTE *mo, CODIFICATION *cod, SYMBOL *mi, FILE_SIZE s, SIZE sym) {
	unsigned int start, blocks;
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	blocks = s / TOTAL_THREADS;
	blocks = tid < (s % TOTAL_THREADS) ? blocks+1 : blocks;

	start = (s / TOTAL_THREADS) * tid;
	start = tid >= (s % TOTAL_THREADS) ? start + (s % TOTAL_THREADS) : start + tid;

	BYTE buffer = 0;
	wb[tid] = 0;

	unsigned int i;
	for(i = 0 ; i < blocks ; i++) {
		unsigned int j;
		for(j = 0 ; j < sym ; j++) {
			if(mi[start + i] == cod[j].symbol) {
				unsigned int tam_string;
				for(tam_string = 0 ; cod[j].code[tam_string] != '\0' ; i++);
				tam_string--;
				unsigned int k;
				for(k = 0 ; k < tam_string ; k++) {
					if(cod[j].code[k] == '1') {
						buffer++;
						wb[tid]++;
						if(wb[tid] == 8)
							buffer = 0;
					}
	
					/*if(cod[j].code[k] == '1')
						SET_BIT(mo[tid * s], wb[tid]);
					else
						CLEAR_BIT(mo[tid * s], wb[tid]);

					wb[tid]++;*/
				}
				break;
			}
		}
	}
}

void memoryToFile (void) {
	BYTE buffer;
	unsigned short int bits = 0;

	unsigned int i;
	for(i = 0 ; i < TOTAL_THREADS ; i++) {
		unsigned int j;
		for(j = 0 ; j < written_bits[i] ; j++) {
			if(GET_BIT(memory_out[i], j))
				SET_BIT(&buffer, bits);
			else
				CLEAR_BIT(&buffer, bits);

			if(++bits == BYTE_BIT) {
				fwrite(&buffer, 1, 1, out);
				bits = 0;
			}
		}
	}
	if(bits)
		fwrite(&buffer, 1, 1, out);
}














/* huffman.c */

/* FUNÇÕES ENCODER E DECODER */
void openFiles (FILE **a, char *path_a, char *arg_a, FILE **b, char *path_b, char *arg_b, FILE **c, char *path_c, char *arg_c) {
	*a = fopen(path_a, arg_a);
	if(*a == NULL) {
		fprintf(STDOUT, "Erro ao abrir o arquivo %s\n", path_a);
		exit(0);
	}
	*b = fopen(path_b, arg_b);
	if(*b == NULL) {
		fprintf(STDOUT, "Erro ao abrir o arquivo %s\n", path_b);
		exit(0);
	}
	*c = fopen(path_c, arg_c);
	if(*c == NULL) {
		fprintf(STDOUT, "Erro ao abrir o arquivo %s\n", path_c);
		exit(0);
	}
}

unsigned int strlenU(unsigned char * str) {
	unsigned int i;
	for(i = 0 ; (*(str++)) ; i++);
	return i;
}

void newNode (NODE *node, SYMBOL symbol, FREQUENCY frequency, NODE *left, NODE *right) {
	node->symbol = symbol;
	node->frequency = frequency;
	if(left) {
		node->left = (NODE *) malloc(sizeof(NODE));
		*(node->left) = *left;
	}
	else
		node->left = NULL;
	if(right) {
		node->right = (NODE *) malloc(sizeof(NODE));
		*(node->right) = *right;
	}
	else
		node->right = NULL;

	node->visited = 0;
}

NODE_ARRAY * newNodeArray (SIZE size) {
	NODE_ARRAY *a = (NODE_ARRAY *) malloc(sizeof(NODE_ARRAY));
	a->node = (NODE *) malloc(sizeof(NODE) * size);
	a->size = 0;

	return a;
}

void insertSorted (NODE_ARRAY *a, NODE *n) {
	SIZE i, j;
	for(i = 0 ; i < a->size && a->node[i].frequency > n->frequency ; i++);
	a->size++;
	for(j = a->size ; j > i ; j--)
		a->node[j] = a->node[j-1];

	a->node[j] = *n;
}

void removeLastNodes (NODE_ARRAY *a, SIZE num) {
	a->size -= num;
}

STACK * newStack (POWER_SIZE size) {
	STACK *s = (STACK *) malloc(sizeof(STACK));
	s->stack = (NODE *) malloc(sizeof(NODE) * size);
	s->top = 0;

	return s;
}

NODE * pop (STACK *s) {
	NODE *n = &(s->stack[s->top-1]);
	s->top--;

	return n;
}

void push (STACK *s, NODE *n) {
	s->stack[s->top] = *n;
	s->top++;
}

unsigned int power (unsigned int a, unsigned int b) {
	unsigned int resultado = 1;

	unsigned int i;
	for (i = 0 ; i < b ; i++)
		resultado *= a;

	return resultado;
}

void divideArray (unsigned int size, unsigned int tid, unsigned int *start, unsigned int *blocks) {
	*blocks = size / BLOCKS_PER_GRID * THREADS_PER_BLOCK;
	*blocks = tid < (size % NTHREADS) ? *blocks+1 : *blocks;

	*start = (size / NTHREADS) * tid;
	*start = tid >= (size % NTHREADS) ? *start + (size % NTHREADS) : *start + tid;
}


/* FUNÇÕES ENCODER */
SYMBOL * fileToMemory(FILE *file, FILE_SIZE *size) {
	SYMBOL *memory;
	fseek (file , 0 , SEEK_END);
	*size = ftell (file);
	rewind (file);

	memory = (SYMBOL *) malloc(*size + (SIZEOF_SYMBOL) + 10);
	if(!memory) {
		fprintf(STDOUT, "Erro ao alocar espaço na memória para o arquivo\n");
		return NULL;
	}

	fread(memory, *size, 1, file);
	return memory;
}

void huffmanEncode (NODE_ARRAY *array) {
	while(array->size > 1) {
		NODE *a, *b, *c;
		a = &(array->node[array->size-2]);
		b = &(array->node[array->size-1]);
		c = (NODE *) malloc(sizeof(NODE));
		newNode(c, 0, a->frequency + b->frequency, a, b);
		//fprintf(stderr, "%u(%u) e %u(%u)  --->   %u(%u)\n", a->symbol, a->frequency, b->symbol, b->frequency, c->symbol, c->frequency);
		removeLastNodes(array, 2);
		insertSorted(array, c);
	}
}

void codificationToFile(FILE *file, CODIFICATION *codification, SIZE symbols, SYMBOL eof) {
	SIZE eof_index = 0;

	SIZE i;
	for(i = 0 ; i < symbols ; i++) {
		if(codification[i].symbol != eof) {
			//fprintf(STDOUT, "%u %u %s\n", codification[i].symbol, codification[i].size, codification[i].code);
			fwrite(&codification[i].symbol, SIZEOF_SYMBOL, 1, file);
			fwrite(&codification[i].size, SIZEOF_SYMBOL, 1, file);
			fwrite(codification[i].code, sizeof(unsigned char), codification[i].size, file);
		}
		else
			eof_index = i;
	}
	//fprintf(STDOUT, "%u %u %s\n", codification[eof_index].symbol, codification[eof_index].size, codification[eof_index].code);
	fwrite(&codification[eof_index].symbol, SIZEOF_SYMBOL, 1, file);
	fwrite(&codification[eof_index].size, SIZEOF_SYMBOL, 1, file);
	fwrite(codification[eof_index].code, sizeof(unsigned char), codification[eof_index].size, file);
}

/* FUNÇÕES DECODER */
CODIFICATION * fileToCode (FILE *file, SIZE *symbols, unsigned int *max_code) {
	CODIFICATION *codification;

	unsigned int read = 1;
	*symbols = 0;
	*max_code = 0;
	codification = (CODIFICATION *) malloc(sizeof(CODIFICATION) * POWER_BITS_CODIFICATION);
	do {
		fread(&codification[*symbols].symbol, SIZEOF_SYMBOL, 1, file);
		fread(&codification[*symbols].size, SIZEOF_SYMBOL, 1, file);
		codification[*symbols].code = (unsigned char *) malloc(sizeof(unsigned char) * (codification[*symbols].size+1));
		read = fread(codification[*symbols].code, sizeof(unsigned char), codification[*symbols].size, file);
		if(codification[*symbols].size > *max_code)
			*max_code = codification[*symbols].size;
		(*symbols)++;
	} while(read);
	(*symbols)--;
	return codification;
}

void huffmanDecode(FILE *in, FILE *out, CODIFICATION_ARRAY_ELEMENT *treeArray, SYMBOL eof) {
	uint8_t feof = 0;
	unsigned int index = 0;

	unsigned int i;
	for(i = 0 ; !feof ; i++) {
		BYTE buffer;
		fread(&buffer, 1, 1, in);
		unsigned int j;
		for(j = 0 ; j < BYTE_BIT ; j++) {
			index <<= 1;
			if((GET_BIT(&buffer, j)) == 0)
				index += 1;
			else
				index += 2;

			if(treeArray[index].used) {
				if(treeArray[index].symbol == eof) {
					feof = 1;
					return;
				}
				else
					fwrite(&treeArray[index].symbol, SIZEOF_SYMBOL, 1, out);
				index = 0;
			}
		}
	}
}
